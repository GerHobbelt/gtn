#include <sstream>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

#include "hip/hip_runtime.h"

namespace gtn {

namespace {

void add(const float* a, const float* b, float* out, size_t size) {
  thrust::device_ptr<const float> aPtr(a);
  thrust::device_ptr<const float> bPtr(b);
  thrust::device_ptr<float> outPtr(out);
  thrust::transform(aPtr, aPtr + size, bPtr, outPtr, thrust::plus<float>());
}

void copyDeviceDevice(int* dst, const int* src, size_t size) {
  CUDA_CHECK(hipMemcpyAsync(
    static_cast<void*>(dst),
    static_cast<const void*>(src),
    size * sizeof(int),
    hipMemcpyDefault));
}

void copyHostDevice(int* dst, const std::vector<int>& src, size_t size) {
  CUDA_CHECK(hipMemcpyAsync(
    static_cast<void*>(dst),
    static_cast<const void*>(src.data()),
    size * sizeof(int),
    hipMemcpyDefault));
}

void copyDeviceHost(std::vector<int>& dst, const int* src, size_t size) {
  dst.resize(size);
  CUDA_CHECK(hipMemcpy(
    static_cast<void*>(dst.data()),
    static_cast<const void*>(src),
    size * sizeof(int),
    hipMemcpyDefault));
}

} // namespace

Graph Graph::cpu() const {

  // No-op if already on CPU
  if (!sharedGraph_->isCuda) {
    return *this;
  }
  Graph g;
  auto& hd = *(g.sharedGraph_);
  hd.isCuda = false;
  hd.compiled = true;
  g.setCalcGrad(this->calcGrad());
  auto& dd = this->sharedGraph_->deviceData;
  hd.numNodes = dd.numNodes;
  hd.numArcs = dd.numArcs;
  copyDeviceHost(hd.start, dd.start, g.numNodes());
  copyDeviceHost(hd.accept, dd.accept, g.numNodes());
  copyDeviceHost(hd.inArcOffset, dd.inArcOffset, g.numNodes() + 1);
  copyDeviceHost(hd.outArcOffset, dd.outArcOffset, g.numNodes() + 1);
  copyDeviceHost(hd.inArcs, dd.inArcs, g.numArcs());
  copyDeviceHost(hd.outArcs, dd.outArcs, g.numArcs());
  copyDeviceHost(hd.ilabels, dd.ilabels, g.numArcs());
  copyDeviceHost(hd.olabels, dd.olabels, g.numArcs());
  copyDeviceHost(hd.srcNodes, dd.srcNodes, g.numArcs());
  copyDeviceHost(hd.dstNodes, dd.dstNodes, g.numArcs());
  // Get the indices of the start and accept nodes
  for (int i = 0; i < hd.start.size(); i++) {
    if (hd.start[i]) {
      hd.startIds.push_back(i);
    }
  }
  for (int i = 0; i < hd.accept.size(); i++) {
    if (hd.accept[i]) {
      hd.acceptIds.push_back(i);
    }
  }

  g.sharedWeights_->weights.resize(g.numArcs());
  CUDA_CHECK(hipMemcpy(
    static_cast<void*>(g.weights()),
    static_cast<const void*>(this->weights()),
    g.numArcs() * sizeof(int),
    hipMemcpyDefault));
  return g;
}

Graph Graph::cuda(int device_) const {
  // No-op if already on GPU
  if (isCuda() && device() == device_) {
    return *this;
  }
  maybeCompile();

  Graph g;
  g.sharedGraph_->startIds = this->start();
  g.sharedGraph_->acceptIds = this->accept();
  g.sharedGraph_->isCuda = true;
  g.sharedGraph_->compiled = true;
  g.sharedGraph_->numNodes = this->numNodes();
  g.sharedGraph_->numArcs = this->numArcs(); 
  g.setCalcGrad(this->calcGrad());
  g.sharedGraph_->device = device_;

  auto& hd = *(this->sharedGraph_);
  auto& dd = g.sharedGraph_->deviceData;
  cuda::detail::DeviceManager dm(device_);
  if (!isCuda()) {
    dd.allocate(this->numNodes(), this->numArcs());
    copyHostDevice(dd.start, hd.start, g.numNodes());
    copyHostDevice(dd.accept, hd.accept, g.numNodes());
    copyHostDevice(dd.inArcOffset, hd.inArcOffset, g.numNodes() + 1);
    copyHostDevice(dd.outArcOffset, hd.outArcOffset, g.numNodes() + 1);
    copyHostDevice(dd.inArcs, hd.inArcs, g.numArcs());
    copyHostDevice(dd.outArcs, hd.outArcs, g.numArcs());
    copyHostDevice(dd.ilabels, hd.ilabels, g.numArcs());
    copyHostDevice(dd.olabels, hd.olabels, g.numArcs());
    copyHostDevice(dd.srcNodes, hd.srcNodes, g.numArcs());
    copyHostDevice(dd.dstNodes, hd.dstNodes, g.numArcs());
    CUDA_CHECK(hipMalloc(
      (void**)(&g.sharedWeights_->deviceWeights),
      g.numArcs() * sizeof(float)));
    CUDA_CHECK(hipMemcpyAsync(
      static_cast<void*>(g.weights()),
      static_cast<const void*>(this->weights()),
      g.numArcs() * sizeof(float),
      hipMemcpyDefault));
  } else {
    dd.deepCopy(hd.deviceData, device_);
  }
  return g;
}

Graph Graph::cuda() const {
  return cuda(cuda::getDevice());
}

void Graph::addGrad(const float* other) {
  if (!isCuda()) {
    throw std::logic_error(
      "[Graph::addGrad] This addGrad is only for GPU graphs.");
  }
  if (calcGrad()) {
    std::lock_guard<std::mutex> lock(sharedGraph_->grad_lock);
    if (isGradAvailable()) {
      add(other, grad().weights(), grad().weights(), numArcs());
    } else {
      sharedGrad_->grad = std::make_unique<Graph>(false);
      sharedGrad_->grad->sharedGraph_ = sharedGraph_;
      sharedGrad_->grad->sharedWeights_->deepCopy(other, numArcs(), device());
    }
  }
}

void Graph::GraphGPU::allocate(size_t numNodes, size_t numArcs) {
  this->numNodes = numNodes;
  this->numArcs = numArcs;
  CUDA_CHECK(hipMalloc((void**)(&start), sizeof(int) * numNodes));
  CUDA_CHECK(hipMalloc((void**)(&accept), sizeof(int) * numNodes));
  CUDA_CHECK(hipMalloc((void**)(&inArcOffset), sizeof(int) * (numNodes + 1)));
  CUDA_CHECK(hipMalloc((void**)(&outArcOffset), sizeof(int) * (numNodes + 1)));
  CUDA_CHECK(hipMalloc((void**)(&inArcs), sizeof(int) * numArcs));
  CUDA_CHECK(hipMalloc((void**)(&outArcs), sizeof(int) * numArcs));
  CUDA_CHECK(hipMalloc((void**)(&ilabels), sizeof(int) * numArcs));
  CUDA_CHECK(hipMalloc((void**)(&olabels), sizeof(int) * numArcs));
  CUDA_CHECK(hipMalloc((void**)(&srcNodes), sizeof(int) * numArcs));
  CUDA_CHECK(hipMalloc((void**)(&dstNodes), sizeof(int) * numArcs));
}

void Graph::GraphGPU::deepCopy(
    const Graph::GraphGPU& other, int device) {
  cuda::detail::DeviceManager dm(device);
  allocate(other.numNodes, other.numArcs);
  copyDeviceDevice(start, other.start, numNodes);
  copyDeviceDevice(accept, other.accept, numNodes);
  copyDeviceDevice(inArcOffset, other.inArcOffset, numNodes + 1);
  copyDeviceDevice(outArcOffset, other.outArcOffset, numNodes + 1);
  copyDeviceDevice(inArcs, other.inArcs, numArcs);
  copyDeviceDevice(outArcs, other.outArcs, numArcs);
  copyDeviceDevice(ilabels, other.ilabels, numArcs);
  copyDeviceDevice(olabels, other.olabels, numArcs);
  copyDeviceDevice(srcNodes, other.srcNodes, numArcs);
  copyDeviceDevice(dstNodes, other.dstNodes, numArcs);
}

void Graph::GraphGPU::free() {
  if (start != nullptr) {
    CUDA_CHECK(hipFree(start)); 
    CUDA_CHECK(hipFree(accept)); 
    CUDA_CHECK(hipFree(inArcOffset)); 
    CUDA_CHECK(hipFree(outArcOffset)); 
    CUDA_CHECK(hipFree(inArcs)); 
    CUDA_CHECK(hipFree(outArcs)); 
    CUDA_CHECK(hipFree(ilabels)); 
    CUDA_CHECK(hipFree(olabels)); 
    CUDA_CHECK(hipFree(srcNodes)); 
    CUDA_CHECK(hipFree(dstNodes)); 
  }
}

void Graph::SharedWeights::deepCopy(
    const float *src, size_t numArcs, int device) {
  cuda::detail::DeviceManager dm(device);
  CUDA_CHECK(hipMalloc((void**)(&deviceWeights), sizeof(float) * numArcs));
  CUDA_CHECK(hipMemcpyAsync(
    static_cast<void*>(deviceWeights),
    static_cast<const void*>(src),
    numArcs * sizeof(float),
    hipMemcpyDefault));
}

Graph::SharedWeights::~SharedWeights() {
  if (deviceWeights != nullptr) {
    CUDA_CHECK(hipFree(deviceWeights));
  }
}

namespace cuda {

bool isAvailable() {
  return deviceCount() > 0;
}

int deviceCount() {
  int count;
  CUDA_CHECK(hipGetDeviceCount(&count));
  return count;
}

int getDevice() {
  int device;
  CUDA_CHECK(hipGetDevice(&device));
  return device;
}

void setDevice(int device) {
  CUDA_CHECK(hipSetDevice(device));
}

namespace detail {

float* ones(size_t size, int device) {
  DeviceManager dm(device);
  float *res;
  CUDA_CHECK(hipMalloc((void**)(&res), size * sizeof(float)));
  thrust::device_ptr<float> dPtr(res);
  thrust::fill(dPtr, dPtr + size, 1.0f);
  return res;
}

void free(float* ptr) {
  CUDA_CHECK(hipFree(static_cast<void*>(ptr)));
}

void cudaCheck(hipError_t err, const char* file, int line) {
  if (err != hipSuccess) {
    std::ostringstream ess;
    ess << '[' << file << ':' << line
        << "] CUDA error: " << hipGetErrorString(err);
    throw std::runtime_error(ess.str());
  }
}

} // namespace detail
} // namespace cuda
} // namespace gtn
