#include <sstream>
#include <iostream>

#include "hip/hip_runtime.h"

namespace gtn {

namespace {

void copyDeviceDevice(int* dst, const int* src, size_t size) {
  CUDA_CHECK(hipMemcpyAsync(
    static_cast<void*>(dst),
    static_cast<const void*>(src),
    size * sizeof(int),
    hipMemcpyDefault));
}

void copyHostDevice(int* dst, const std::vector<int>& src, size_t size) {
  CUDA_CHECK(hipMemcpyAsync(
    static_cast<void*>(dst),
    static_cast<const void*>(src.data()),
    size * sizeof(int),
    hipMemcpyDefault));
}

void copyDeviceHost(std::vector<int>& dst, const int* src, size_t size) {
  dst.resize(size);
  CUDA_CHECK(hipMemcpy(
    static_cast<void*>(dst.data()),
    static_cast<const void*>(src),
    size * sizeof(int),
    hipMemcpyDefault));
}

} // namespace

Graph Graph::cpu() {
  // No-op if already on CPU
  if (!sharedGraph_->isCuda) {
    return *this;
  }
  Graph g;
  g.sharedGraph_->startIds = this->start();
  g.sharedGraph_->acceptIds = this->accept();
  g.sharedGraph_->isCuda = false;
  g.sharedGraph_->compiled = true;
  g.sharedGraph_->numNodes = this->numNodes();
  g.sharedGraph_->numArcs = this->numArcs();
  g.setCalcGrad(this->calcGrad());
  auto& dd = this->sharedGraph_->deviceData;
  auto& hd = *(g.sharedGraph_);
  copyDeviceHost(hd.start, dd.start, g.numNodes());
  copyDeviceHost(hd.accept, dd.accept, g.numNodes());
  copyDeviceHost(hd.inArcOffset, dd.inArcOffset, g.numNodes() + 1);
  copyDeviceHost(hd.outArcOffset, dd.outArcOffset, g.numNodes() + 1);
  copyDeviceHost(hd.inArcs, dd.inArcs, g.numArcs());
  copyDeviceHost(hd.outArcs, dd.outArcs, g.numArcs());
  copyDeviceHost(hd.ilabels, dd.ilabels, g.numArcs());
  copyDeviceHost(hd.olabels, dd.olabels, g.numArcs());
  copyDeviceHost(hd.srcNodes, dd.srcNodes, g.numArcs());
  copyDeviceHost(hd.dstNodes, dd.dstNodes, g.numArcs());

  g.sharedWeights_->resize(g.numArcs());
  CUDA_CHECK(hipMemcpy(
    static_cast<void*>(g.weights()),
    static_cast<const void*>(dd.weights),
    g.numArcs() * sizeof(int),
    hipMemcpyDefault));
  return g;
}

Graph Graph::cuda(int device_) {
  // No-op if already on GPU
  if (isCuda() && device() == device_) {
    return *this;
  }
  maybeCompile();

  Graph g;
  g.sharedGraph_->startIds = this->start();
  g.sharedGraph_->acceptIds = this->accept();
  g.sharedGraph_->isCuda = true;
  g.sharedGraph_->compiled = true;
  g.sharedGraph_->numNodes = this->numNodes();
  g.sharedGraph_->numArcs = this->numArcs(); 
  g.setCalcGrad(this->calcGrad());
  g.sharedGraph_->device = device_;

  auto& hd = *(this->sharedGraph_);
  auto& dd = g.sharedGraph_->deviceData;
  cuda::detail::DeviceManager dm(device_);
  dd.allocate(g.numNodes(), g.numArcs());
  if (!isCuda()) {
    copyHostDevice(dd.start, hd.start, g.numNodes());
    copyHostDevice(dd.accept, hd.accept, g.numNodes());
    copyHostDevice(dd.inArcOffset, hd.inArcOffset, g.numNodes() + 1);
    copyHostDevice(dd.outArcOffset, hd.outArcOffset, g.numNodes() + 1);
    copyHostDevice(dd.inArcs, hd.inArcs, g.numArcs());
    copyHostDevice(dd.outArcs, hd.outArcs, g.numArcs());
    copyHostDevice(dd.ilabels, hd.ilabels, g.numArcs());
    copyHostDevice(dd.olabels, hd.olabels, g.numArcs());
    copyHostDevice(dd.srcNodes, hd.srcNodes, g.numArcs());
    copyHostDevice(dd.dstNodes, hd.dstNodes, g.numArcs());
    CUDA_CHECK(hipMemcpyAsync(
      static_cast<void*>(dd.weights),
      static_cast<const void*>(this->weights()),
      g.numArcs() * sizeof(float),
      hipMemcpyDefault));
  } else {
    dd.deepCopy(hd.deviceData, g.numNodes(), g.numArcs(), device_);
  }
  return g;
}

Graph Graph::cuda() {
  return cuda(cuda::getDevice());
}

void Graph::GraphGPU::allocate(size_t numNodes, size_t numArcs) {
  CUDA_CHECK(hipMalloc((void**)(&start), sizeof(int) * numNodes));
  CUDA_CHECK(hipMalloc((void**)(&accept), sizeof(int) * numNodes));
  CUDA_CHECK(hipMalloc((void**)(&inArcOffset), sizeof(int) * (numNodes + 1)));
  CUDA_CHECK(hipMalloc((void**)(&outArcOffset), sizeof(int) * (numNodes + 1)));
  CUDA_CHECK(hipMalloc((void**)(&inArcs), sizeof(int) * numArcs));
  CUDA_CHECK(hipMalloc((void**)(&outArcs), sizeof(int) * numArcs));
  CUDA_CHECK(hipMalloc((void**)(&ilabels), sizeof(int) * numArcs));
  CUDA_CHECK(hipMalloc((void**)(&olabels), sizeof(int) * numArcs));
  CUDA_CHECK(hipMalloc((void**)(&srcNodes), sizeof(int) * numArcs));
  CUDA_CHECK(hipMalloc((void**)(&dstNodes), sizeof(int) * numArcs));
  CUDA_CHECK(hipMalloc((void**)(&weights), sizeof(float) * numArcs));
}

void Graph::GraphGPU::deepCopy(
    const Graph::GraphGPU& other, size_t numNodes, size_t numArcs, int device) {
  cuda::detail::DeviceManager dm(device);
  allocate(numNodes, numArcs);
  copyDeviceDevice(start, other.start, numNodes);
  copyDeviceDevice(accept, other.accept, numNodes);
  copyDeviceDevice(inArcOffset, other.inArcOffset, numNodes + 1);
  copyDeviceDevice(outArcOffset, other.outArcOffset, numNodes + 1);
  copyDeviceDevice(inArcs, other.inArcs, numArcs);
  copyDeviceDevice(outArcs, other.outArcs, numArcs);
  copyDeviceDevice(ilabels, other.ilabels, numArcs);
  copyDeviceDevice(olabels, other.olabels, numArcs);
  copyDeviceDevice(srcNodes, other.srcNodes, numArcs);
  copyDeviceDevice(dstNodes, other.dstNodes, numArcs);
  CUDA_CHECK(hipMemcpyAsync(
    static_cast<void*>(weights),
    static_cast<const void*>(other.weights),
    numArcs * sizeof(float),
    hipMemcpyDeviceToDevice));
}

Graph::GraphGPU::~GraphGPU() {
  if (start != nullptr) {
    CUDA_CHECK(hipFree(start)); 
    CUDA_CHECK(hipFree(accept)); 
    CUDA_CHECK(hipFree(inArcOffset)); 
    CUDA_CHECK(hipFree(outArcOffset)); 
    CUDA_CHECK(hipFree(inArcs)); 
    CUDA_CHECK(hipFree(outArcs)); 
    CUDA_CHECK(hipFree(ilabels)); 
    CUDA_CHECK(hipFree(olabels)); 
    CUDA_CHECK(hipFree(srcNodes)); 
    CUDA_CHECK(hipFree(dstNodes)); 
  }
}

namespace cuda {

bool isAvailable() {
  return deviceCount() > 0;
}

int deviceCount() {
  int count;
  CUDA_CHECK(hipGetDeviceCount(&count));
  return count;
}

int getDevice() {
  int device;
  CUDA_CHECK(hipGetDevice(&device));
  return device;
}

void setDevice(int device) {
  CUDA_CHECK(hipSetDevice(device));
}

namespace detail {

void cudaCheck(hipError_t err, const char* file, int line) {
  if (err != hipSuccess) {
    std::ostringstream ess;
    ess << '[' << file << ':' << line
        << "] CUDA error: " << hipGetErrorString(err);
    throw std::runtime_error(ess.str());
  }
}

} // namespace detail
} // namespace cuda
} // namespace gtn
