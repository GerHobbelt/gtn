#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

#include "hip/hip_runtime.h"

namespace gtn {

namespace {

void copy(int* dst, const int* src, size_t size) {
  CUDA_CHECK(hipMemcpyAsync(
    static_cast<void*>(dst),
    static_cast<const void*>(src),
    size * sizeof(int),
    hipMemcpyDefault));
}

} // namespace

void Graph::SharedGraph::allocDevice() {
  cuda::detail::DeviceManager dm(device);
  CUDA_CHECK(hipMalloc((void**)(&startIds), sizeof(int) * numNodes));
  CUDA_CHECK(hipMalloc((void**)(&acceptIds), sizeof(int) * numNodes));
  CUDA_CHECK(hipMalloc((void**)(&start), sizeof(int) * numNodes));
  CUDA_CHECK(hipMalloc((void**)(&accept), sizeof(int) * numNodes));
  CUDA_CHECK(hipMalloc((void**)(&inArcOffset), sizeof(int) * (numNodes + 1)));
  CUDA_CHECK(hipMalloc((void**)(&outArcOffset), sizeof(int) * (numNodes + 1)));
  CUDA_CHECK(hipMalloc((void**)(&inArcs), sizeof(int) * numArcs));
  CUDA_CHECK(hipMalloc((void**)(&outArcs), sizeof(int) * numArcs));
  CUDA_CHECK(hipMalloc((void**)(&ilabels), sizeof(int) * numArcs));
  CUDA_CHECK(hipMalloc((void**)(&olabels), sizeof(int) * numArcs));
  CUDA_CHECK(hipMalloc((void**)(&srcNodes), sizeof(int) * numArcs));
  CUDA_CHECK(hipMalloc((void**)(&dstNodes), sizeof(int) * numArcs));
}

void Graph::SharedGraph::deepCopy(const Graph::SharedGraph& other) {
  numNodes = other.numNodes;
  numArcs = other.numArcs; 
  numStart = other.numStart;
  numAccept = other.numAccept;
  if (isCuda) {
    allocDevice();
  } else {
    allocHost();
  }
  copy(startIds, other.startIds, numStart);
  copy(acceptIds, other.acceptIds, numAccept);
  copy(start, other.start, numNodes);
  copy(accept, other.accept, numNodes);
  copy(inArcOffset, other.inArcOffset, numNodes + 1);
  copy(outArcOffset, other.outArcOffset, numNodes + 1);
  copy(inArcs, other.inArcs, numArcs);
  copy(outArcs, other.outArcs, numArcs);
  copy(ilabels, other.ilabels, numArcs);
  copy(olabels, other.olabels, numArcs);
  copy(srcNodes, other.srcNodes, numArcs);
  copy(dstNodes, other.dstNodes, numArcs);
}

void Graph::SharedGraph::freeDevice() {
  CUDA_CHECK(hipFree(startIds)); 
  CUDA_CHECK(hipFree(acceptIds)); 
  CUDA_CHECK(hipFree(start)); 
  CUDA_CHECK(hipFree(accept)); 
  CUDA_CHECK(hipFree(inArcOffset)); 
  CUDA_CHECK(hipFree(outArcOffset)); 
  CUDA_CHECK(hipFree(inArcs)); 
  CUDA_CHECK(hipFree(outArcs)); 
  CUDA_CHECK(hipFree(ilabels)); 
  CUDA_CHECK(hipFree(olabels)); 
  CUDA_CHECK(hipFree(srcNodes)); 
  CUDA_CHECK(hipFree(dstNodes)); 
}

void Graph::SharedWeights::allocDevice(size_t numArcs, int device) {
  isCuda = true;
  cuda::detail::DeviceManager dm(device);
  CUDA_CHECK(hipMalloc((void**)(&weights), sizeof(float) * numArcs));
}

void Graph::SharedWeights::deepCopy(
    const float *src, size_t numArcs, bool isCuda, int device) {
  if (isCuda) {
    allocDevice(numArcs, device);
  } else {
    allocHost(numArcs);
  }
  CUDA_CHECK(hipMemcpyAsync(
    static_cast<void*>(weights),
    static_cast<const void*>(src),
    numArcs * sizeof(float),
    hipMemcpyDefault));
}

} // namespace gtn
